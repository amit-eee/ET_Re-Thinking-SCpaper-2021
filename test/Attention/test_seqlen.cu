#include "hip/hip_runtime.h"
#include <Attention/Attention.h>
#include <CUDA_ptr.hpp>
#include <Linear/Linear.h>
#include <Model.h>
#include <bits/stdc++.h>
#include <wtime.h>
using namespace culib;
using w_mat_t = base_mat;
using mat_v_t = w_mat_t;

template <typename _attn_T> void test() {
    using attn_config =
        Attention_config<w_mat_t, w_mat_t, mat_v_t, w_mat_t, _attn_T>;
    constexpr float sparsity = 0.6;
    std::vector<double> res;
    for (int seq_len = 16; seq_len <= 512; seq_len += 16) {
        auto para = std::make_shared<Model_t>(
            Model_t{768, 768, seq_len, 12, 4096, 768});
        CUDA_ptr<half> IN(para->emdim * para->seq_len, __float2half_rn(0.2));
        CUDA_ptr<half> d_OUT(para->emdim * para->seq_len);
        auto LQ = gen_sparse_linear<w_mat_t>(para->kdim, para->emdim,
                                             para->seq_len, sparsity);
        auto LK = gen_sparse_linear<w_mat_t>(para->kdim, para->emdim,
                                             para->seq_len, sparsity);
        auto LV = gen_sparse_linear<mat_v_t>(para->vdim, para->emdim,
                                             para->seq_len, sparsity);
        auto LO = gen_sparse_linear<w_mat_t>(para->emdim, para->vdim,
                                             para->seq_len, sparsity);
        auto attn = std::make_unique<Attention<attn_config>>(
            std::move(LQ), std::move(LK), std::move(LV), std::move(LO), para);
        attn->forward(d_OUT.get(), IN.get(), IN.get(), IN.get());
        cudaChk(hipDeviceSynchronize());
        auto time = wtime(
            10,
            [&]() {
                attn->forward(d_OUT.get(), IN.get(), IN.get(), IN.get());
                hipDeviceSynchronize();
            },
            []() {});
        d_OUT.clear();
        res.push_back(time);
    }
    std::cout << typeid(_attn_T).name() << std::endl;
    for (auto i : res) {
        printf("%lf\n", i);
    }
}

int main(int ac, char **av) {
    test<OTF_attn_full>();
    test<OTF_attn_sharedQK>();
}
