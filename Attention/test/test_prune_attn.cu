#include "hip/hip_runtime.h"
#include "../../Linear/Linear.h"
#include "../Attention.h"
#include <CUDA_ptr.hpp>
#include <bits/stdc++.h>
#include <wtime.h>
using namespace culib;

int main(int ac, char** av) {
    float sparsity = std::atof(av[1]);
    using w_mat_t = base_mat;
    using vw_mat_t = row_mat;
    using attn_config =
        Attention_config<w_mat_t, w_mat_t, vw_mat_t, w_mat_t, Prune_attn>;
    auto para =
        std::make_shared<Model_t>(Model_t{768, 768, 384, 12, 3072, 768});

    auto LQ = gen_sparse_linear<w_mat_t>(para->kdim, para->emdim, para->seq_len, sparsity);
    auto LK = gen_sparse_linear<w_mat_t>(para->kdim, para->emdim, para->seq_len, sparsity);
    auto LV = gen_sparse_linear<vw_mat_t>(para->vdim, para->emdim, para->seq_len, sparsity);
    auto LO = gen_sparse_linear<w_mat_t>(para->emdim, para->vdim, para->seq_len, sparsity);
    auto attn = std::make_unique<Attention<attn_config>>(
        std::move(LQ), std::move(LK), std::move(LV), std::move(LO), para);

    CUDA_ptr<half> IN(para->emdim * para->seq_len);
    CUDA_ptr<half> d_OUT(para->emdim * para->seq_len);
    attn->forward(d_OUT.get(), IN.get(), IN.get(), IN.get());
    hipDeviceSynchronize();

    auto time = wtime(
        10,
        [&]() {
            attn->forward(d_OUT.get(), IN.get(), IN.get(), IN.get());
            hipDeviceSynchronize();
        },
        []() {});
    std::cout << "Time: " << time << " us\n";

    std::vector<half> OUT(para->emdim * para->seq_len);
    d_OUT.dump(OUT.data());
    std::cout << __half2float(OUT[0]) << std::endl;
}