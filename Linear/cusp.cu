#include "Linear.h"
#include <cstdio>
#define CHECK_CUDA(func)                                                       \
    {                                                                          \
        hipError_t status = (func);                                           \
        if (status != hipSuccess) {                                           \
            printf("CUDA API failed at line %d with error: %s (%d)\n",         \
                   __LINE__, hipGetErrorString(status), status);              \
            return EXIT_FAILURE;                                               \
        }                                                                      \
    }

#define CHECK_CUSPARSE(func)                                                   \
    {                                                                          \
        hipsparseStatus_t status = (func);                                      \
        if (status != HIPSPARSE_STATUS_SUCCESS) {                               \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n",     \
                   __LINE__, hipsparseGetErrorString(status), status);          \
            return EXIT_FAILURE;                                               \
        }                                                                      \
    }

void Linear<csr_mat>::forward(half *output, half *input,
                              hipStream_t stream) {
    hipsparseSetStream(handle, stream);
    
    auto bias_temp = this->bias.get();
    auto stride = out_size;
    const auto add_bias = [bias_temp, stride] __device__(half * data,
                                                         int i) -> half {
        return data[i] + bias_temp[i % stride];
    }; // unpruned bias
    static const half alpha = half_one, beta = half_zero;
    hipsparseCreateDnMat(&denmat, in_size, size, in_size, input, HIP_R_16F,
                        HIPSPARSE_ORDER_COL);
    hipsparseCreateDnMat(&resmat, out_size, size, out_size, output, HIP_R_16F,
                        HIPSPARSE_ORDER_COL);
    size_t bufferSize = 0;
    hipsparseSpMM_bufferSize(handle, opA, opB, &alpha, spWeight, denmat, &beta,
                            resmat, HIP_R_16F, HIPSPARSE_SPMM_CSR_ALG2,
                            &bufferSize);
    culib::CUDA_ptr<half> workspace(bufferSize);

    hipsparseSpMM(handle, opA, opB, &alpha, spWeight, denmat, &beta, resmat,
                 HIP_R_16F, HIPSPARSE_SPMM_CSR_ALG2, workspace.get());

    culib::cuda_map(output, size * out_size, add_bias, stream);
    hipsparseDestroyDnMat(denmat);
    hipsparseDestroyDnMat(resmat);
}

Linear<csr_mat>::~Linear() {
    hipsparseDestroy(handle);
    hipsparseDestroySpMat(spWeight);
    delete indptr;
    delete indices;
    delete data;
}

Linear<csr_mat>::Linear(int _in_size, int _out_size, const csr_mat &w,
                        const half *b, int _size)
    : bias(b, _out_size), in_size(_in_size), out_size(_out_size), size(_size) {
    hipsparseCreate(&handle);
    indptr = new culib::CUDA_ptr<int>(w.indptr);
    indices = new culib::CUDA_ptr<int>(w.indices);
    data = new culib::CUDA_ptr<half>(w.data.size());
    culib::CUDA_ptr<float> data_f(w.data);
    culib::util::to_half_devptr(data->get(), data_f.get(), w.data.size());
    hipsparseCreateCsr(&spWeight, w.nrow, w.ncol, w.nnz, indptr->get(),
                      indices->get(), data->get(), HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F);
}

Linear<csr_mat>::Linear(Linear<csr_mat> &&_linear)
    : in_size(_linear.in_size), out_size(_linear.out_size), size(_linear.size),
      bias(std::move(_linear.bias)), indptr(_linear.indptr),
      indices(_linear.indices), data(_linear.data) {
    hipsparseCreate(&handle);
    hipsparseCreateCsr(&spWeight, out_size, in_size, data->size, indptr->get(),
                      indices->get(), data->get(), HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F);
}
