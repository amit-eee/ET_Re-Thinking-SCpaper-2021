#include "hip/hip_runtime.h"
#include "mixed_attn.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <mma.hpp>
namespace cg = cooperative_groups;

__global__ void __kernel_multi_head_full_mixed(
    const half *__restrict__ Q, const half *__restrict__ K,
    const half *__restrict__ V, half *__restrict__ Z, const int kdim,
    const int vdim, const int seq_len, const int num_head,
    const half *__restrict__ mask) {
    // blockIdx.x: block row id
    // blockIdx.y: head_id
    using frag_t = culib::mma::mma_t<16, 16, 16>;
    // S_row_f + (Q_row, S_row)
    extern __shared__ char smem[];
    auto cta = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(cta);
    const auto warp_id = warp.meta_group_rank();
    const auto lane_id = warp.thread_rank();
    const auto num_warp = warp.meta_group_size();
    const auto head_dim = kdim / num_head;
    auto Q_ptr = &Q[16 * blockIdx.x * kdim + head_dim * blockIdx.y];
    auto temp_Q = reinterpret_cast<half *>(&smem[sizeof(float) * 16 * seq_len]);
    for (int r = warp_id; r < 16; r += num_warp) {
        auto dst = &temp_Q[r * head_dim];
        auto src = &Q_ptr[r * kdim];
        cg::memcpy_async(warp, dst, src, sizeof(half) * head_dim);
    }
    cta.sync();
    frag_t::a_t<wmma::row_major> a_frag;
    frag_t::b_t<wmma::col_major> b_frag;
    frag_t::c_t<float> c_frag;
    auto S_row_f = reinterpret_cast<float *>(smem);
    for (int KR = warp_id; KR < seq_len / 16; KR += num_warp) {
        auto K_ptr = &K[16 * KR * kdim + head_dim * blockIdx.y];
        wmma::fill_fragment(c_frag, 0);
        for (int i = 0; i < head_dim; i += 16) {
            wmma::load_matrix_sync(a_frag, temp_Q + i, head_dim);
            wmma::load_matrix_sync(b_frag, K_ptr + i, kdim);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
        wmma::store_matrix_sync(S_row_f + KR * 16, c_frag, seq_len,
                                wmma::mem_row_major);
    }
    cta.sync();
    // scale
    const half scale = hsqrt(__int2half_rn(head_dim));
    auto temp_row = temp_Q; // S_row_h
    for (int i = cta.thread_rank(); i < seq_len * 16; i += cta.size()) {
        const auto temp = __float2half_rn(S_row_f[i]);
        temp_row[i] = __hdiv(temp, scale);
    }
    cta.sync();
    // mask
    const auto mask_base =
        reinterpret_cast<const half2 *>(&mask[(blockIdx.x * 16) * seq_len]);
    auto temp_row_2 = reinterpret_cast<half2 *>(temp_row);
    for (int i = threadIdx.x; i < seq_len * 8; i += blockDim.x) {
        temp_row_2[i] += mask_base[i];
    }
    cta.sync();
    for (int row = warp_id; row < 16; row += num_warp) {
        auto row_ptr = temp_row + row * seq_len;
        // find the max
        half val_max = half_zero, temp;
        for (auto i = warp.thread_rank(); i < seq_len; i += warp.size()) {
            temp = row_ptr[i];
            val_max = val_max > temp ? val_max : temp;
        }
        warp.sync();
        const auto max = cg::reduce(warp, val_max, cg::greater<half>());
        // compute the sum of exp-ed and shifted array
        half val_sum = half_zero;
        for (auto i = warp.thread_rank(); i < seq_len; i += warp.size()) {
            temp = hexp(row_ptr[i] - max);
            val_sum += temp;
            row_ptr[i] = temp;
        }
        warp.sync();
        const auto sum = cg::reduce(warp, val_sum, cg::plus<half>());
        // update with softmax scaling
        for (auto i = warp.thread_rank(); i < seq_len; i += warp.size()) {
            row_ptr[i] = row_ptr[i] / sum;
        }
    }
    cta.sync();
    const auto vhead_dim = vdim / num_head;
    for (int VC = warp_id; VC < vhead_dim / 16; VC += num_warp) {
        frag_t::b_t<wmma::row_major> b_frag;
        frag_t::c_t<half> c_frag;
        wmma::fill_fragment(c_frag, half_zero);
        for (int i = 0; i < seq_len; i += 16) {
            auto V_ptr = &V[vdim * i + blockIdx.y * vhead_dim + VC * 16];
            wmma::load_matrix_sync(a_frag, temp_row + i, seq_len);
            wmma::load_matrix_sync(b_frag, V_ptr, vdim);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
        auto res =
            &Z[blockIdx.x * 16 * vdim + blockIdx.y * vhead_dim + VC * 16];
        wmma::store_matrix_sync(res, c_frag, vdim, wmma::mem_row_major);
    }
}

template <>
void Multihead_atttion<OTF_attn_full_mixed>(ptr_t *mat_q, ptr_t *mat_k,
                                            ptr_t *mat_v, ptr_t *mat_qk,
                                            const ptr_t *mask, ptr_t *mat_z,
                                            std::shared_ptr<Model_t> model) {
    const auto Q_row = sizeof(half) * 16 * (model->kdim / model->nhead);
    const auto S_row_f = sizeof(float) * 16 * model->seq_len;
    const auto S_row_h = sizeof(half) * 16 * model->seq_len;
    const auto smem_size = std::max(Q_row, S_row_h) + S_row_f;
    int num_thd, _num_blk;
    cudaChk(hipOccupancyMaxPotentialBlockSize(
        &_num_blk, &num_thd, __kernel_multi_head_full_mixed, smem_size));
    const auto num_blk = dim3(model->seq_len / 16, model->nhead);
    __kernel_multi_head_full_mixed<<<num_blk, num_thd, smem_size>>>(
        mat_q->get(), mat_k->get(), mat_v->get(), mat_z->get(), model->kdim,
        model->vdim, model->seq_len, model->nhead, mask->get());
}